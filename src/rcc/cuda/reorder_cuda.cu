#include "hip/hip_runtime.h"
/*
 *  EXP-T -- A Relativistic Fock-Space Multireference Coupled Cluster Program
 *  Copyright (C) 2018-2023 The EXP-T developers.
 *
 *  This file is part of EXP-T.
 *
 *  EXP-T is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  EXP-T is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with EXP-T.  If not, see <http://www.gnu.org/licenses/>.
 *
 *  E-mail:        exp-t-program@googlegroups.com
 *  Google Groups: https://groups.google.com/d/forum/exp-t-program
 */

/**
 * Multidimensional tensor transposition on GPU.
 * Current implementation can work only with 4-dim tensors
 * (extension is rather straightforward)
 */

#include "cuda_code.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "engine.h"

extern "C" {

// must be descreased for Fermi (comp cap = 2.x)
#define BLOCK_SIZE 1024

// constant memory
__constant__ int64_t
        d_nsize;
__constant__ int32_t
        d_perm[CC_DIAGRAM_MAX_RANK];
__constant__ int64_t
        d_coef1[CC_DIAGRAM_MAX_RANK];
__constant__ int64_t
        d_coef2[CC_DIAGRAM_MAX_RANK];

/*******************************************************************************
 * reorder_kernel_rank4
 *
 * reorder -- kernel function for 4-dim tensors.
 * instructions are grouped in order to have as less register variables
 * as possible (in order to avoid calls to the local memory storage).
 * this can be of extreme importance for old architectures (Tesla, Fermi).
 * Note that the cached constant memory is intensively used.
 *
 * Do not try to understand anything, better see reorder.c (the same algorithm,
 * but with normal number of variables, much more readable).
 ******************************************************************************/
__global__ void reorder_kernel_rank4_complex(hipDoubleComplex *v1, hipDoubleComplex *v2)
{
    __shared__
    int16_t idx4[4 * BLOCK_SIZE];
    int64_t coef;

    int64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= d_nsize) { return; }

    int64_t offs = 4 * threadIdx.x;

    // dim 0
    coef = d_coef1[0];
    idx4[offs + 0] = i / coef;
    i = i % coef;

    // dim 1
    coef = d_coef1[1];
    idx4[offs + 1] = i / coef;
    i = i % coef;

    // dim 2
    coef = d_coef1[2];
    idx4[offs + 2] = i / coef;
    i = i % coef;

    // dim 3
    idx4[offs + 3] = i;

    // use 'coef' variable instead of 'index2' variable
    coef = d_coef2[0] * idx4[offs + d_perm[0]] + d_coef2[1] * idx4[offs + d_perm[1]] +
           d_coef2[2] * idx4[offs + d_perm[2]] + idx4[offs + d_perm[3]];

    // copy matrix element from global memory to global
    i = threadIdx.x + blockIdx.x * blockDim.x;
    v2[coef /* index2 */] = v1[i];
}

__global__ void reorder_kernel_rank4_real(double *v1, double *v2)
{
    __shared__
    int16_t idx4[4 * BLOCK_SIZE];
    int64_t coef;

    int64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= d_nsize) { return; }

    int64_t offs = 4 * threadIdx.x;

    // dim 0
    coef = d_coef1[0];
    idx4[offs + 0] = i / coef;
    i = i % coef;

    // dim 1
    coef = d_coef1[1];
    idx4[offs + 1] = i / coef;
    i = i % coef;

    // dim 2
    coef = d_coef1[2];
    idx4[offs + 2] = i / coef;
    i = i % coef;

    // dim 3
    idx4[offs + 3] = i;

    // use 'coef' variable instead of 'index2' variable
    coef = d_coef2[0] * idx4[offs + d_perm[0]] + d_coef2[1] * idx4[offs + d_perm[1]] +
           d_coef2[2] * idx4[offs + d_perm[2]] + idx4[offs + d_perm[3]];

    // copy matrix element from global memory to global
    i = threadIdx.x + blockIdx.x * blockDim.x;
    v2[coef /* index2 */] = v1[i];
}

int reorder_cuda(int carith, int rank, int *perm, int *dims1, int *dims2, double _Complex *v1, double _Complex *v2)
{
    int64_t coef1[CC_DIAGRAM_MAX_RANK];
    int64_t coef2[CC_DIAGRAM_MAX_RANK];
    int64_t nsize;
    //int block_size = 32;//engine_config.n_cuda_threads;
    int i, j, idim;
    hipError_t cuerr;
    hipDoubleComplex *d_v1, *d_v2;
    int SZ_WORKING_TYPE = carith ? sizeof(double _Complex) : sizeof(double);

    if (rank != 4) {
        fprintf(stderr, "reorder on GPU is now available only for 4-dimensional tensors\n");
        fprintf(stderr, "please, contact alexvoleynichenko@mail.com the new extended code is needed\n");
        fprintf(stderr, "see %s for details\n", __FILE__);
        return 1;
    }

    // calculate total number of elements to be reordered
    nsize = 1;
    for (idim = 0; idim < rank; idim++) {
        nsize *= dims1[idim];
    }

    // prepare coefficients for recalculation: compound index <-> linear index
    for (i = 0; i < rank; i++) {
        coef1[i] = 1;
        coef2[i] = 1;
    }
    for (i = 0; i < rank - 1; i++) {
        coef1[i] = 1;
        coef2[i] = 1;
        for (j = i + 1; j < rank; j++) {
            coef1[i] *= dims1[j];
            coef2[i] *= dims2[j];
        }
    }

    // copy all constant data to the constant memory
    cuerr = hipMemcpyToSymbol(HIP_SYMBOL(d_nsize), &nsize, sizeof(int64_t) * 1, 0, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy 'nsize' from host to device: %s\n", hipGetErrorString(cuerr));
        return 1;
    }
    cuerr = hipMemcpyToSymbol(HIP_SYMBOL(d_perm), perm, sizeof(int32_t) * rank, 0, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy 'perm' from host to device: %s\n", hipGetErrorString(cuerr));
        return 1;
    }
    cuerr = hipMemcpyToSymbol(HIP_SYMBOL(d_coef1), coef1, sizeof(int64_t) * rank, 0, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy 'coef1' from host to device: %s\n", hipGetErrorString(cuerr));
        return 1;
    }
    cuerr = hipMemcpyToSymbol(HIP_SYMBOL(d_coef2), coef2, sizeof(int64_t) * rank, 0, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy 'coef2' from host to device: %s\n", hipGetErrorString(cuerr));
        return 1;
    }

    // alloc arrays on device
    cuerr = hipMalloc((void **) &d_v1, nsize * SZ_WORKING_TYPE);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot allocate GPU memory for d_v1: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }
    cuerr = hipMalloc((void **) &d_v2, nsize * SZ_WORKING_TYPE);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot allocate GPU memory for d_v2: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }

    // copy v1 to GPU
    cuerr = hipMemcpy(d_v1, v1, nsize * SZ_WORKING_TYPE, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy data (v1) from host to device: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }

    // launch kernel
    dim3 threads = dim3(BLOCK_SIZE, 1, 1);
    dim3 blocks = dim3((nsize + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);

    if (carith) {
        reorder_kernel_rank4_complex <<< blocks, threads >>>(d_v1, d_v2);
    }
    else {
        reorder_kernel_rank4_real <<< blocks, threads >>>((double *) d_v1, (double *) d_v2);
    }
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot launch kernel: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }

    cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot synchronize device: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }

    // get result (v2) from GPU
    cuerr = hipMemcpy(v2, d_v2, nsize * SZ_WORKING_TYPE, hipMemcpyDeviceToHost);
    if (cuerr != hipSuccess) {
        fprintf(stderr, "Cannot copy result (v2) from device to host: %s\n",
                hipGetErrorString(cuerr));
        fflush(stderr);
        return 1;
    }

    hipFree(d_v1);
    hipFree(d_v2);

    return 0;
}

} /* extern "C" */
